#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2020 Neka-Nat
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 * 
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
**/
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/iterator/constant_iterator.h>

#include "cupoch/geometry/boundingvolume.h"
#include "cupoch/geometry/graph.h"
#include "cupoch/geometry/lineset.h"
#include "cupoch/geometry/pointcloud.h"
#include "cupoch/geometry/trianglemesh.h"
#include "cupoch/geometry/distancetransform.h"
#include "cupoch/geometry/geometry_functor.h"
#include "cupoch/geometry/voxelgrid.h"
#include "cupoch/utility/platform.h"
#include "cupoch/utility/range.h"
#include "cupoch/visualization/shader/shader.h"
#include "cupoch/visualization/shader/simple_shader.h"
#include "cupoch/visualization/utility/color_map.h"
#include "cupoch/visualization/visualizer/render_option.h"

using namespace cupoch;
using namespace cupoch::visualization;
using namespace cupoch::visualization::glsl;

namespace {

// Vertex indices of 12 lines in a cuboid
__constant__ int cuboid_lines_vertex_indices[12][2] = {
        {0, 1}, {0, 2}, {0, 4}, {3, 1}, {3, 2}, {3, 7},
        {5, 1}, {5, 4}, {5, 7}, {6, 2}, {6, 4}, {6, 7},
};

template <int Dim>
struct copy_pointcloud_functor {
    copy_pointcloud_functor(bool has_colors,
                            RenderOption::PointColorOption color_option,
                            const ViewControl &view)
        : has_colors_(has_colors), color_option_(color_option), view_(view){};
    const bool has_colors_;
    const RenderOption::PointColorOption color_option_;
    const ViewControl view_;
    const ColorMap::ColorMapOption colormap_option_ = GetGlobalColorMapOption();
    __device__ thrust::tuple<Eigen::Vector3f, Eigen::Vector4f> operator()(
            const thrust::tuple<Eigen::Matrix<float, Dim, 1>, Eigen::Vector3f> &pt_cl);

    __device__ Eigen::Vector4f GetColor(const Eigen::Vector3f& point,
                                        const Eigen::Vector3f& color) const {
        Eigen::Vector4f color_tmp;
        color_tmp[3] = 1.0;
        switch (color_option_) {
            case RenderOption::PointColorOption::XCoordinate:
                color_tmp.head<3>() = GetColorMapColor(
                        view_.GetBoundingBox().GetXPercentage(point(0)),
                        colormap_option_);
                break;
            case RenderOption::PointColorOption::YCoordinate:
                color_tmp.head<3>() = GetColorMapColor(
                        view_.GetBoundingBox().GetYPercentage(point(1)),
                        colormap_option_);
                break;
            case RenderOption::PointColorOption::ZCoordinate:
                color_tmp.head<3>() = GetColorMapColor(
                        view_.GetBoundingBox().GetZPercentage(point(2)),
                        colormap_option_);
                break;
            case RenderOption::PointColorOption::Color:
            case RenderOption::PointColorOption::Default:
            default:
                if (has_colors_) {
                    color_tmp.head<3>() = color;
                } else {
                    color_tmp.head<3>() = GetColorMapColor(
                            view_.GetBoundingBox().GetZPercentage(point(2)),
                            colormap_option_);
                }
                break;
        }
        return color_tmp;
    }
};

template <>
__device__
thrust::tuple<Eigen::Vector3f, Eigen::Vector4f> copy_pointcloud_functor<3>::operator()(
            const thrust::tuple<Eigen::Vector3f, Eigen::Vector3f> &pt_cl) {
    const Eigen::Vector3f &point = thrust::get<0>(pt_cl);
    const Eigen::Vector3f &color = thrust::get<1>(pt_cl);
    return thrust::make_tuple(point, GetColor(point, color));
}

template <>
__device__
thrust::tuple<Eigen::Vector3f, Eigen::Vector4f> copy_pointcloud_functor<2>::operator()(
            const thrust::tuple<Eigen::Vector2f, Eigen::Vector3f> &pt_cl) {
    const Eigen::Vector3f point = (Eigen::Vector3f() << thrust::get<0>(pt_cl), 0.0).finished();
    const Eigen::Vector3f &color = thrust::get<1>(pt_cl);
    return thrust::make_tuple(point, GetColor(point, color));
}


struct copy_lineset_functor {
    copy_lineset_functor(
            const thrust::pair<Eigen::Vector3f, Eigen::Vector3f> *line_coords,
            const Eigen::Vector3f *line_colors,
            bool has_colors)
        : line_coords_(line_coords),
          line_colors_(line_colors),
          has_colors_(has_colors){};
    const thrust::pair<Eigen::Vector3f, Eigen::Vector3f> *line_coords_;
    const Eigen::Vector3f *line_colors_;
    const bool has_colors_;
    const Eigen::Vector3f default_line_color_ = geometry::DEFAULT_LINE_COLOR;
    __device__ thrust::tuple<Eigen::Vector3f, Eigen::Vector4f> operator()(
            size_t k) const {
        int i = k / 2;
        int j = k % 2;
        Eigen::Vector4f color_tmp;
        color_tmp[3]  = 1.0;
        color_tmp.head<3>() =
                (has_colors_) ? line_colors_[i] : default_line_color_;
        if (j == 0) {
            return thrust::make_tuple(line_coords_[i].first, color_tmp);
        } else {
            return thrust::make_tuple(line_coords_[i].second, color_tmp);
        }
    }
};

template <int Dim>
struct line_coordinates_functor {
    line_coordinates_functor(const Eigen::Matrix<float, Dim, 1> *points) : points_(points){};
    const Eigen::Matrix<float, Dim, 1> *points_;
    __device__ thrust::pair<Eigen::Vector3f, Eigen::Vector3f> operator()(
            const Eigen::Vector2i &idxs) const;
};

template <>
__device__
thrust::pair<Eigen::Vector3f, Eigen::Vector3f> line_coordinates_functor<3>::operator()(
    const Eigen::Vector2i &idxs) const {
    return thrust::make_pair(points_[idxs[0]], points_[idxs[1]]);
}

template <>
__device__
thrust::pair<Eigen::Vector3f, Eigen::Vector3f> line_coordinates_functor<2>::operator()(
    const Eigen::Vector2i &idxs) const {
    const Eigen::Vector3f p1 = (Eigen::Vector3f() << points_[idxs[0]], 0.0).finished();
    const Eigen::Vector3f p2 = (Eigen::Vector3f() << points_[idxs[1]], 0.0).finished();
    return thrust::make_pair(p1, p2);
}

struct copy_trianglemesh_functor {
    copy_trianglemesh_functor(const Eigen::Vector3f *vertices,
                              const int *triangles,
                              const Eigen::Vector3f *vertex_colors,
                              bool has_vertex_colors,
                              RenderOption::MeshColorOption color_option,
                              const Eigen::Vector3f &default_mesh_color,
                              const ViewControl &view)
        : vertices_(vertices),
          triangles_(triangles),
          vertex_colors_(vertex_colors),
          has_vertex_colors_(has_vertex_colors),
          color_option_(color_option),
          default_mesh_color_(default_mesh_color),
          view_(view){};
    const Eigen::Vector3f *vertices_;
    const int *triangles_;
    const Eigen::Vector3f *vertex_colors_;
    const bool has_vertex_colors_;
    const RenderOption::MeshColorOption color_option_;
    const Eigen::Vector3f default_mesh_color_;
    const ViewControl view_;
    const ColorMap::ColorMapOption colormap_option_ = GetGlobalColorMapOption();
    __device__ thrust::tuple<Eigen::Vector3f, Eigen::Vector4f> operator()(
            size_t k) const {
        size_t vi = triangles_[k];
        const auto &vertex = vertices_[vi];
        Eigen::Vector4f color_tmp;
        color_tmp[3] = 1.0;
        switch (color_option_) {
            case RenderOption::MeshColorOption::XCoordinate:
                color_tmp.head<3>() = GetColorMapColor(
                        view_.GetBoundingBox().GetXPercentage(vertex(0)),
                        colormap_option_);
                break;
            case RenderOption::MeshColorOption::YCoordinate:
                color_tmp.head<3>() = GetColorMapColor(
                        view_.GetBoundingBox().GetYPercentage(vertex(1)),
                        colormap_option_);
                break;
            case RenderOption::MeshColorOption::ZCoordinate:
                color_tmp.head<3>() = GetColorMapColor(
                        view_.GetBoundingBox().GetZPercentage(vertex(2)),
                        colormap_option_);
                break;
            case RenderOption::MeshColorOption::Color:
                if (has_vertex_colors_) {
                    color_tmp.head<3>() = vertex_colors_[vi];
                    break;
                }
            case RenderOption::MeshColorOption::Default:
            default:
                color_tmp.head<3>() = default_mesh_color_;
                break;
        }
        return thrust::make_tuple(vertex, color_tmp);
    }
};

struct copy_voxelgrid_line_functor {
    copy_voxelgrid_line_functor(const Eigen::Vector3f *vertices,
                                const geometry::Voxel *voxels,
                                bool has_colors,
                                RenderOption::MeshColorOption color_option,
                                const Eigen::Vector3f &default_mesh_color,
                                const ViewControl &view)
        : vertices_(vertices),
          voxels_(voxels),
          has_colors_(has_colors),
          color_option_(color_option),
          default_mesh_color_(default_mesh_color),
          view_(view){};
    const Eigen::Vector3f *vertices_;
    const geometry::Voxel *voxels_;
    const bool has_colors_;
    const RenderOption::MeshColorOption color_option_;
    const Eigen::Vector3f default_mesh_color_;
    const ViewControl view_;
    const ColorMap::ColorMapOption colormap_option_ = GetGlobalColorMapOption();
    __device__ thrust::tuple<Eigen::Vector3f, Eigen::Vector4f> operator()(
            size_t idx) const {
        int i = idx / (12 * 2);
        int jk = idx % (12 * 2);
        int j = jk / 2;
        int k = jk % 2;
        // Voxel color (applied to all points)
        Eigen::Vector4f voxel_color;
        voxel_color[3] = 1.0;
        switch (color_option_) {
            case RenderOption::MeshColorOption::XCoordinate:
                voxel_color.head<3>() =
                        GetColorMapColor(view_.GetBoundingBox().GetXPercentage(
                                                 vertices_[i * 8](0)),
                                         colormap_option_);
                break;
            case RenderOption::MeshColorOption::YCoordinate:
                voxel_color.head<3>() =
                        GetColorMapColor(view_.GetBoundingBox().GetYPercentage(
                                                 vertices_[i * 8](1)),
                                         colormap_option_);
                break;
            case RenderOption::MeshColorOption::ZCoordinate:
                voxel_color.head<3>() =
                        GetColorMapColor(view_.GetBoundingBox().GetZPercentage(
                                                 vertices_[i * 8](2)),
                                         colormap_option_);
                break;
            case RenderOption::MeshColorOption::Color:
                if (has_colors_) {
                    voxel_color.head<3>() = voxels_[i].color_;
                    break;
                }
            case RenderOption::MeshColorOption::Default:
            default:
                voxel_color.head<3>() = default_mesh_color_;
                break;
        }
        return thrust::make_tuple(
                vertices_[i * 8 + cuboid_lines_vertex_indices[j][k]],
                voxel_color);
    }
};

struct copy_distance_voxel_functor {
    copy_distance_voxel_functor(float voxel_size,
                                int resolution,
                                const Eigen::Vector3f& origin,
                                float distance_max)
        : voxel_size_(voxel_size), resolution_(resolution),
        origin_(origin), distance_max_(distance_max){};
    const float voxel_size_;
    const int resolution_;
    const Eigen::Vector3f origin_;
    const float distance_max_;
    __device__ thrust::tuple<Eigen::Vector3f, Eigen::Vector4f>
    operator()(const thrust::tuple<size_t, geometry::DistanceVoxel>& kv) const {
        int idx = thrust::get<0>(kv);
        geometry::DistanceVoxel v = thrust::get<1>(kv);
        int res2 = resolution_ * resolution_;
        int x = idx / res2;
        int yz = idx % res2;
        int y = yz / resolution_;
        int z = yz % resolution_;
        // Voxel color (applied to all points)
        Eigen::Vector4f voxel_color = Eigen::Vector4f::Ones();
        int h_res = resolution_ / 2;
        Eigen::Vector3f pt = (Eigen::Vector3i(x - h_res, y - h_res, z - h_res).cast<float>() + Eigen::Vector3f::Constant(0.5)) * voxel_size_ - origin_;
        voxel_color[3] = 1.0 - min(v.distance_, distance_max_) / distance_max_;
        return thrust::make_tuple(pt, voxel_color);
    }
};

struct alpha_greater_functor {
    __device__ bool operator() (const thrust::tuple<Eigen::Vector3f, Eigen::Vector4f>& lhs,
                                const thrust::tuple<Eigen::Vector3f, Eigen::Vector4f>& rhs) const {
        return thrust::get<1>(lhs)[3] > thrust::get<1>(rhs)[3];
    }
};

}  // namespace

bool SimpleShader::Compile() {
    if (CompileShaders(simple_vertex_shader, NULL, simple_fragment_shader) ==
        false) {
        PrintShaderWarning("Compiling shaders failed.");
        return false;
    }
    vertex_position_ = glGetAttribLocation(program_, "vertex_position");
    vertex_color_ = glGetAttribLocation(program_, "vertex_color");
    MVP_ = glGetUniformLocation(program_, "MVP");
    return true;
}

void SimpleShader::Release() {
    UnbindGeometry(true);
    ReleaseProgram();
}

bool SimpleShader::BindGeometry(const geometry::Geometry &geometry,
                                const RenderOption &option,
                                const ViewControl &view) {
    // If there is already geometry, we first unbind it.
    // We use GL_STATIC_DRAW. When geometry changes, we clear buffers and
    // rebind the geometry. Note that this approach is slow. If the geometry is
    // changing per frame, consider implementing a new ShaderWrapper using
    // GL_STREAM_DRAW, and replace InvalidateGeometry() with Buffer Object
    // Streaming mechanisms.
    UnbindGeometry();

    // Prepare data to be passed to GPU
    const size_t num_data_size = GetDataSize(geometry);

    // Create buffers and bind the geometry
    glGenBuffers(1, &vertex_position_buffer_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_position_buffer_);
    glBufferData(GL_ARRAY_BUFFER, num_data_size * sizeof(Eigen::Vector3f), 0,
                 GL_STATIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    cudaSafeCall(hipGraphicsGLRegisterBuffer(&cuda_graphics_resources_[0],
                                              vertex_position_buffer_,
                                              cudaGraphicsMapFlagsNone));
    glGenBuffers(1, &vertex_color_buffer_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_color_buffer_);
    glBufferData(GL_ARRAY_BUFFER, num_data_size * sizeof(Eigen::Vector4f), 0,
                 GL_STATIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    cudaSafeCall(hipGraphicsGLRegisterBuffer(&cuda_graphics_resources_[1],
                                              vertex_color_buffer_,
                                              cudaGraphicsMapFlagsNone));

    Eigen::Vector3f *raw_points_ptr;
    Eigen::Vector4f *raw_colors_ptr;
    size_t n_bytes;
    cudaSafeCall(hipGraphicsMapResources(2, cuda_graphics_resources_));
    cudaSafeCall(hipGraphicsResourceGetMappedPointer(
            (void **)&raw_points_ptr, &n_bytes, cuda_graphics_resources_[0]));
    cudaSafeCall(hipGraphicsResourceGetMappedPointer(
            (void **)&raw_colors_ptr, &n_bytes, cuda_graphics_resources_[1]));
    thrust::device_ptr<Eigen::Vector3f> dev_points_ptr =
            thrust::device_pointer_cast(raw_points_ptr);
    thrust::device_ptr<Eigen::Vector4f> dev_colors_ptr =
            thrust::device_pointer_cast(raw_colors_ptr);

    if (PrepareBinding(geometry, option, view, dev_points_ptr,
                       dev_colors_ptr) == false) {
        PrintShaderWarning("Binding failed when preparing data.");
        return false;
    }

    Unmap(2);
    bound_ = true;
    return true;
}

bool SimpleShader::RenderGeometry(const geometry::Geometry &geometry,
                                  const RenderOption &option,
                                  const ViewControl &view) {
    if (PrepareRendering(geometry, option, view) == false) {
        PrintShaderWarning("Rendering failed during preparation.");
        return false;
    }
    glUseProgram(program_);
    glUniformMatrix4fv(MVP_, 1, GL_FALSE, view.GetMVPMatrix().data());
    glEnableVertexAttribArray(vertex_position_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_position_buffer_);
    glVertexAttribPointer(vertex_position_, 3, GL_FLOAT, GL_FALSE, 0, NULL);
    glEnableVertexAttribArray(vertex_color_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_color_buffer_);
    glVertexAttribPointer(vertex_color_, 4, GL_FLOAT, GL_FALSE, 0, NULL);
    glDrawArrays(draw_arrays_mode_, 0, draw_arrays_size_);
    glDisableVertexAttribArray(vertex_position_);
    glDisableVertexAttribArray(vertex_color_);
    return true;
}

void SimpleShader::UnbindGeometry(bool finalize) {
    if (bound_) {
        if (!finalize) {
            cudaSafeCall(hipGraphicsUnregisterResource(
                    cuda_graphics_resources_[0]));
            cudaSafeCall(hipGraphicsUnregisterResource(
                    cuda_graphics_resources_[1]));
        }
        glDeleteBuffers(1, &vertex_position_buffer_);
        glDeleteBuffers(1, &vertex_color_buffer_);
        bound_ = false;
    }
}

bool SimpleShaderForPointCloud::PrepareRendering(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view) {
    if (geometry.GetGeometryType() !=
        geometry::Geometry::GeometryType::PointCloud) {
        PrintShaderWarning("Rendering type is not geometry::PointCloud.");
        return false;
    }
    glPointSize(GLfloat(option.point_size_));
    glEnable(GL_DEPTH_TEST);
    glDepthFunc(GLenum(option.GetGLDepthFunc()));
    return true;
}

bool SimpleShaderForPointCloud::PrepareBinding(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view,
        thrust::device_ptr<Eigen::Vector3f> &points,
        thrust::device_ptr<Eigen::Vector4f> &colors) {
    if (geometry.GetGeometryType() !=
        geometry::Geometry::GeometryType::PointCloud) {
        PrintShaderWarning("Rendering type is not geometry::PointCloud.");
        return false;
    }
    const geometry::PointCloud &pointcloud =
            (const geometry::PointCloud &)geometry;
    if (pointcloud.HasPoints() == false) {
        PrintShaderWarning("Binding failed with empty pointcloud.");
        return false;
    }
    copy_pointcloud_functor<3> func(pointcloud.HasColors(),
                                    option.point_color_option_, view);
    if (pointcloud.HasColors()) {
        thrust::transform(
                make_tuple_begin(pointcloud.points_, pointcloud.colors_),
                make_tuple_end(pointcloud.points_, pointcloud.colors_),
                make_tuple_iterator(points, colors), func);
    } else {
        thrust::transform(
                make_tuple_iterator(pointcloud.points_.begin(),
                                    thrust::constant_iterator<Eigen::Vector3f>(
                                            Eigen::Vector3f::Zero())),
                make_tuple_iterator(pointcloud.points_.end(),
                                    thrust::constant_iterator<Eigen::Vector3f>(
                                            Eigen::Vector3f::Zero())),
                make_tuple_iterator(points, colors), func);
    }
    draw_arrays_mode_ = GL_POINTS;
    draw_arrays_size_ = GLsizei(pointcloud.points_.size());
    return true;
}

size_t SimpleShaderForPointCloud::GetDataSize(
        const geometry::Geometry &geometry) const {
    return ((const geometry::PointCloud &)geometry).points_.size();
}

bool SimpleShaderForLineSet::PrepareRendering(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view) {
    if (geometry.GetGeometryType() !=
        geometry::Geometry::GeometryType::LineSet) {
        PrintShaderWarning("Rendering type is not geometry::LineSet.");
        return false;
    }
    glLineWidth(GLfloat(option.line_width_));
    glEnable(GL_DEPTH_TEST);
    glDepthFunc(GLenum(option.GetGLDepthFunc()));
    return true;
}

bool SimpleShaderForLineSet::PrepareBinding(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view,
        thrust::device_ptr<Eigen::Vector3f> &points,
        thrust::device_ptr<Eigen::Vector4f> &colors) {
    if (geometry.GetGeometryType() !=
        geometry::Geometry::GeometryType::LineSet) {
        PrintShaderWarning("Rendering type is not geometry::LineSet.");
        return false;
    }
    const geometry::LineSet<3> &lineset =
            (const geometry::LineSet<3> &)geometry;
    if (lineset.HasLines() == false) {
        PrintShaderWarning("Binding failed with empty geometry::LineSet.");
        return false;
    }
    utility::device_vector<thrust::pair<Eigen::Vector3f, Eigen::Vector3f>>
            line_coords(lineset.lines_.size());
    line_coordinates_functor<3> func_line(
            thrust::raw_pointer_cast(lineset.points_.data()));
    thrust::transform(lineset.lines_.begin(), lineset.lines_.end(),
                      line_coords.begin(), func_line);
    copy_lineset_functor func_cp(
            thrust::raw_pointer_cast(line_coords.data()),
            thrust::raw_pointer_cast(lineset.colors_.data()),
            lineset.HasColors());
    thrust::transform(thrust::make_counting_iterator<size_t>(0),
                      thrust::make_counting_iterator(lineset.lines_.size() * 2),
                      make_tuple_iterator(points, colors), func_cp);
    draw_arrays_mode_ = GL_LINES;
    draw_arrays_size_ = GLsizei(lineset.lines_.size() * 2);
    return true;
}

size_t SimpleShaderForLineSet::GetDataSize(
        const geometry::Geometry &geometry) const {
    return ((const geometry::LineSet<3> &)geometry).lines_.size() * 2;
}

template <int Dim>
bool SimpleShaderForGraphNode<Dim>::PrepareRendering(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view) {
    if (geometry.GetGeometryType() != geometry::Geometry::GeometryType::Graph) {
        PrintShaderWarning("Rendering type is not geometry::Graph.");
        return false;
    }
    glPointSize(GLfloat(option.point_size_));
    glEnable(GL_DEPTH_TEST);
    glDepthFunc(GLenum(option.GetGLDepthFunc()));
    return true;
}

template <int Dim>
bool SimpleShaderForGraphNode<Dim>::PrepareBinding(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view,
        thrust::device_ptr<Eigen::Vector3f> &points,
        thrust::device_ptr<Eigen::Vector4f> &colors) {
    if (geometry.GetGeometryType() != geometry::Geometry::GeometryType::Graph) {
        PrintShaderWarning("Rendering type is not geometry::Graph.");
        return false;
    }
    const geometry::Graph<Dim> &graph = (const geometry::Graph<Dim> &)geometry;
    if (graph.HasPoints() == false) {
        PrintShaderWarning("Binding failed with empty graph.");
        return false;
    }
    copy_pointcloud_functor<Dim> func(graph.HasColors(), option.point_color_option_,
                                      view);
    if (graph.HasNodeColors()) {
        thrust::transform(make_tuple_begin(graph.points_, graph.node_colors_),
                          make_tuple_end(graph.points_, graph.node_colors_),
                          make_tuple_iterator(points, colors), func);
    } else {
        thrust::transform(
                make_tuple_iterator(graph.points_.begin(),
                                    thrust::constant_iterator<Eigen::Vector3f>(
                                            Eigen::Vector3f::Ones())),
                make_tuple_iterator(graph.points_.end(),
                                    thrust::constant_iterator<Eigen::Vector3f>(
                                            Eigen::Vector3f::Ones())),
                make_tuple_iterator(points, colors), func);
    }
    draw_arrays_mode_ = GL_POINTS;
    draw_arrays_size_ = GLsizei(graph.points_.size());
    return true;
}

template <int Dim>
size_t SimpleShaderForGraphNode<Dim>::GetDataSize(
        const geometry::Geometry &geometry) const {
    return ((const geometry::Graph<Dim> &)geometry).points_.size();
}

template class SimpleShaderForGraphNode<2>;
template class SimpleShaderForGraphNode<3>;

template <int Dim>
bool SimpleShaderForGraphEdge<Dim>::PrepareRendering(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view) {
    if (geometry.GetGeometryType() != geometry::Geometry::GeometryType::Graph) {
        PrintShaderWarning("Rendering type is not geometry::Graph.");
        return false;
    }
    glLineWidth(GLfloat(option.line_width_));
    glEnable(GL_DEPTH_TEST);
    glDepthFunc(GLenum(option.GetGLDepthFunc()));
    return true;
}

template <int Dim>
bool SimpleShaderForGraphEdge<Dim>::PrepareBinding(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view,
        thrust::device_ptr<Eigen::Vector3f> &points,
        thrust::device_ptr<Eigen::Vector4f> &colors) {
    if (geometry.GetGeometryType() != geometry::Geometry::GeometryType::Graph) {
        PrintShaderWarning("Rendering type is not geometry::Graph.");
        return false;
    }
    const geometry::Graph<Dim> &graph = (const geometry::Graph<Dim> &)geometry;
    if (graph.HasLines() == false) {
        PrintShaderWarning("Binding failed with empty geometry::Graph.");
        return false;
    }
    utility::device_vector<thrust::pair<Eigen::Vector3f, Eigen::Vector3f>>
            line_coords(graph.lines_.size());
    line_coordinates_functor<Dim> func_line(
            thrust::raw_pointer_cast(graph.points_.data()));
    thrust::transform(graph.lines_.begin(), graph.lines_.end(),
                      line_coords.begin(), func_line);
    copy_lineset_functor func_cp(thrust::raw_pointer_cast(line_coords.data()),
                                 thrust::raw_pointer_cast(graph.colors_.data()),
                                 graph.HasColors());
    thrust::transform(thrust::make_counting_iterator<size_t>(0),
                      thrust::make_counting_iterator(graph.lines_.size() * 2),
                      make_tuple_iterator(points, colors), func_cp);
    draw_arrays_mode_ = GL_LINES;
    draw_arrays_size_ = GLsizei(graph.lines_.size() * 2);
    return true;
}

template <int Dim>
size_t SimpleShaderForGraphEdge<Dim>::GetDataSize(
        const geometry::Geometry &geometry) const {
    return ((const geometry::Graph<Dim> &)geometry).lines_.size() * 2;
}

template class SimpleShaderForGraphEdge<2>;
template class SimpleShaderForGraphEdge<3>;

bool SimpleShaderForAxisAlignedBoundingBox::PrepareRendering(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view) {
    if (geometry.GetGeometryType() !=
        geometry::Geometry::GeometryType::AxisAlignedBoundingBox) {
        PrintShaderWarning(
                "Rendering type is not geometry::AxisAlignedBoundingBox.");
        return false;
    }
    glLineWidth(GLfloat(option.line_width_));
    glEnable(GL_DEPTH_TEST);
    glDepthFunc(GLenum(option.GetGLDepthFunc()));
    return true;
}

bool SimpleShaderForAxisAlignedBoundingBox::PrepareBinding(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view,
        thrust::device_ptr<Eigen::Vector3f> &points,
        thrust::device_ptr<Eigen::Vector4f> &colors) {
    if (geometry.GetGeometryType() !=
        geometry::Geometry::GeometryType::AxisAlignedBoundingBox) {
        PrintShaderWarning(
                "Rendering type is not geometry::AxisAlignedBoundingBox.");
        return false;
    }
    auto lineset = geometry::LineSet<3>::CreateFromAxisAlignedBoundingBox(
            (const geometry::AxisAlignedBoundingBox<3> &)geometry);
    utility::device_vector<thrust::pair<Eigen::Vector3f, Eigen::Vector3f>>
            line_coords(lineset->lines_.size());
    line_coordinates_functor<3> func_line(
            thrust::raw_pointer_cast(lineset->points_.data()));
    thrust::transform(lineset->lines_.begin(), lineset->lines_.end(),
                      line_coords.begin(), func_line);
    copy_lineset_functor func_cp(
            thrust::raw_pointer_cast(line_coords.data()),
            thrust::raw_pointer_cast(lineset->colors_.data()),
            lineset->HasColors());
    thrust::transform(
            thrust::make_counting_iterator<size_t>(0),
            thrust::make_counting_iterator(lineset->lines_.size() * 2),
            make_tuple_iterator(points, colors), func_cp);
    draw_arrays_mode_ = GL_LINES;
    draw_arrays_size_ = GLsizei(lineset->lines_.size() * 2);
    return true;
}

size_t SimpleShaderForAxisAlignedBoundingBox::GetDataSize(
        const geometry::Geometry &geometry) const {
    auto lineset = geometry::LineSet<3>::CreateFromAxisAlignedBoundingBox(
            (const geometry::AxisAlignedBoundingBox<3> &)geometry);
    return lineset->lines_.size() * 2;
}

bool SimpleShaderForTriangleMesh::PrepareRendering(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view) {
    if (geometry.GetGeometryType() !=
        geometry::Geometry::GeometryType::TriangleMesh) {
        PrintShaderWarning("Rendering type is not geometry::TriangleMesh.");
        return false;
    }
    if (option.mesh_show_back_face_) {
        glDisable(GL_CULL_FACE);
    } else {
        glEnable(GL_CULL_FACE);
    }
    glEnable(GL_DEPTH_TEST);
    glDepthFunc(GLenum(option.GetGLDepthFunc()));
    glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
    if (option.mesh_show_wireframe_) {
        glEnable(GL_POLYGON_OFFSET_FILL);
        glPolygonOffset(1.0, 1.0);
    } else {
        glDisable(GL_POLYGON_OFFSET_FILL);
    }
    return true;
}

bool SimpleShaderForTriangleMesh::PrepareBinding(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view,
        thrust::device_ptr<Eigen::Vector3f> &points,
        thrust::device_ptr<Eigen::Vector4f> &colors) {
    if (geometry.GetGeometryType() !=
        geometry::Geometry::GeometryType::TriangleMesh) {
        PrintShaderWarning("Rendering type is not geometry::TriangleMesh.");
        return false;
    }
    const geometry::TriangleMesh &mesh =
            (const geometry::TriangleMesh &)geometry;
    if (mesh.HasTriangles() == false) {
        PrintShaderWarning("Binding failed with empty triangle mesh.");
        return false;
    }

    copy_trianglemesh_functor func(
            thrust::raw_pointer_cast(mesh.vertices_.data()),
            (int *)(thrust::raw_pointer_cast(mesh.triangles_.data())),
            thrust::raw_pointer_cast(mesh.vertex_colors_.data()),
            mesh.HasVertexColors(), option.mesh_color_option_,
            option.default_mesh_color_, view);
    thrust::transform(
            thrust::make_counting_iterator<size_t>(0),
            thrust::make_counting_iterator(mesh.triangles_.size() * 3),
            make_tuple_iterator(points, colors), func);
    draw_arrays_mode_ = GL_TRIANGLES;
    draw_arrays_size_ = GLsizei(mesh.triangles_.size() * 3);
    return true;
}

size_t SimpleShaderForTriangleMesh::GetDataSize(
        const geometry::Geometry &geometry) const {
    return ((const geometry::TriangleMesh &)geometry).triangles_.size() * 3;
}

bool SimpleShaderForVoxelGridLine::PrepareRendering(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view) {
    if (geometry.GetGeometryType() !=
        geometry::Geometry::GeometryType::VoxelGrid) {
        PrintShaderWarning("Rendering type is not geometry::VoxelGrid.");
        return false;
    }
    glDisable(GL_CULL_FACE);
    glEnable(GL_DEPTH_TEST);
    glDepthFunc(GLenum(option.GetGLDepthFunc()));
    return true;
}

bool SimpleShaderForVoxelGridLine::PrepareBinding(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view,
        thrust::device_ptr<Eigen::Vector3f> &points,
        thrust::device_ptr<Eigen::Vector4f> &colors) {
    if (geometry.GetGeometryType() !=
        geometry::Geometry::GeometryType::VoxelGrid) {
        PrintShaderWarning("Rendering type is not geometry::VoxelGrid.");
        return false;
    }
    const geometry::VoxelGrid &voxel_grid =
            (const geometry::VoxelGrid &)geometry;
    if (voxel_grid.HasVoxels() == false) {
        PrintShaderWarning("Binding failed with empty voxel grid.");
        return false;
    }

    utility::device_vector<Eigen::Vector3f> vertices(
            voxel_grid.voxels_values_.size() * 8);
    thrust::tiled_range<
            thrust::counting_iterator<size_t>>
            irange(thrust::make_counting_iterator<size_t>(0), thrust::make_counting_iterator<size_t>(8),
                   voxel_grid.voxels_values_.size());
    auto gfunc = geometry::get_grid_index_functor<geometry::Voxel, Eigen::Vector3i>();
    auto begin = thrust::make_transform_iterator(voxel_grid.voxels_values_.begin(), gfunc);
    thrust::repeated_range<decltype(begin)>
            vrange(begin, thrust::make_transform_iterator(voxel_grid.voxels_values_.end(), gfunc), 8);
    geometry::compute_voxel_vertices_functor<Eigen::Vector3i> func1(voxel_grid.origin_, voxel_grid.voxel_size_);
    thrust::transform(make_tuple_begin(irange, vrange), make_tuple_end(irange, vrange),
                      vertices.begin(), func1);

    size_t n_out = voxel_grid.voxels_values_.size() * 12 * 2;
    copy_voxelgrid_line_functor func2(
            thrust::raw_pointer_cast(vertices.data()),
            thrust::raw_pointer_cast(voxel_grid.voxels_values_.data()),
            voxel_grid.HasColors(), option.mesh_color_option_,
            option.default_mesh_color_, view);
    thrust::transform(thrust::make_counting_iterator<size_t>(0),
                      thrust::make_counting_iterator(n_out),
                      make_tuple_iterator(points, colors), func2);
    draw_arrays_mode_ = GL_LINES;
    draw_arrays_size_ = GLsizei(n_out);
    return true;
}

size_t SimpleShaderForVoxelGridLine::GetDataSize(
        const geometry::Geometry &geometry) const {
    return ((const geometry::VoxelGrid &)geometry).voxels_values_.size() * 12 *
           2;
}

bool SimpleShaderForDistanceTransform::PrepareRendering(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view) {
    if (geometry.GetGeometryType() !=
        geometry::Geometry::GeometryType::DistanceTransform) {
        PrintShaderWarning("Rendering type is not geometry::DistanceTransform.");
        return false;
    }
    glPointSize(GLfloat(option.point_size_));
    glEnable(GL_DEPTH_TEST);
    glDepthFunc(GLenum(option.GetGLDepthFunc()));
    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
    return true;
}

bool SimpleShaderForDistanceTransform::PrepareBinding(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view,
        thrust::device_ptr<Eigen::Vector3f> &points,
        thrust::device_ptr<Eigen::Vector4f> &colors) {
    if (geometry.GetGeometryType() !=
        geometry::Geometry::GeometryType::DistanceTransform) {
        PrintShaderWarning("Rendering type is not geometry::DistanceTransform.");
        return false;
    }
    const geometry::DistanceTransform &dist_trans =
            (const geometry::DistanceTransform &)geometry;
    if (dist_trans.IsEmpty()) {
        PrintShaderWarning("Binding failed with empty distance transform.");
        return false;
    }

    size_t n_out = dist_trans.voxels_.size();
    copy_distance_voxel_functor
            func(dist_trans.voxel_size_, dist_trans.resolution_, dist_trans.origin_,
                 dist_trans.voxel_size_ * dist_trans.resolution_ * 0.1);
    thrust::transform(make_tuple_iterator(thrust::make_counting_iterator<size_t>(0), dist_trans.voxels_.begin()),
                      make_tuple_iterator(thrust::make_counting_iterator(n_out), dist_trans.voxels_.end()),
                      make_tuple_iterator(points, colors), func);
    auto tp_begin = make_tuple_iterator(points, colors);
    thrust::sort(utility::exec_policy(0),
                 tp_begin, tp_begin + n_out, alpha_greater_functor());
    draw_arrays_mode_ = GL_POINTS;
    draw_arrays_size_ = GLsizei(n_out);
    return true;
}

size_t SimpleShaderForDistanceTransform::GetDataSize(
        const geometry::Geometry &geometry) const {
    int res = ((const geometry::DistanceTransform &)geometry).resolution_;
    return res * res * res;
}
